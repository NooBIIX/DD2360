#include "hip/hip_runtime.h"

#include <stdio.h>
#include <sys/time.h>

#define DataType double
#define TPB 32

// Compute C = A * B
__global__ void gemm(DataType *A, DataType *B, DataType *C, int numARows,
                      int numAColumns, int numBRows, int numBColumns){
  //@@ Insert code to implement matrix multiplication here
  int col_id = blockIdx.x * blockDim.x + threadIdx.x;
  int row_id = blockIdx.y * blockDim.y + threadIdx.y;

  if ((col_id<numBColumns)&&(row_id<numARows)){
    DataType tmp = 0;
    for (int i=0;i<numAColumns;i++){
      tmp += A[numAColumns*row_id+i] * B[numBColumns*i+col_id];
    }
    C[numBColumns*row_id+col_id] = tmp;
  }
}

//@@ Insert code to implement timer start
double timestart(){
  struct timeval t_start;
  gettimeofday(&t_start, NULL);
  return (double) (1000000.0*(t_start.tv_sec) + t_start.tv_usec)/1000000.0;
}
//@@ Insert code to implement timer stop
double timestop(){
  struct timeval t_stop;
  gettimeofday(&t_stop, NULL);
  return (double) (1000000.0*(t_stop.tv_sec) + t_stop.tv_usec)/1000000.0;
}

int main(int argc, char **argv) {
  
  DataType *hostA; // The A matrix
  DataType *hostB; // The B matrix
  DataType *hostC; // The output C matrix
  DataType *resultRef; // The reference result
  DataType *deviceA;
  DataType *deviceB;
  DataType *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;
  int numCColumns;

  //@@ Insert code below to read in numARows, numAColumns, numBColumns from args
  
  numARows = atoi(argv[1]);
  numAColumns = atoi(argv[2]);
  numBRows = atoi(argv[3]);
  numBColumns = atoi(argv[4]);
  numCRows = numARows;
  numCColumns = numBColumns;

  printf("Input matrix dim (%d x %d) (%d x %d) (%d x %d)\n", numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
  
  //@@ Insert code below to allocate Host memory for input and output

  hostA = (DataType*) malloc(numARows * numAColumns * sizeof(DataType));
  hostB = (DataType*) malloc(numBRows * numBColumns * sizeof(DataType));
  hostC = (DataType*) malloc(numCRows * numCColumns * sizeof(DataType));
  resultRef = (DataType*) malloc(numCRows * numCColumns * sizeof(DataType));
  
  //@@ Insert code below to initialize hostA and hostB to random numbers, and create reference result in CPU
  srand((unsigned)time(NULL));
  for(int i=0;i<numARows;i++){
    for (int j=0;j<numAColumns;j++){
      hostA[i*numAColumns+j] = (DataType)rand() / (DataType)RAND_MAX;
    }
  }

    for(int i=0;i<numBRows;i++){
    for (int j=0;j<numBColumns;j++){
      hostB[i*numBColumns+j] = (DataType)rand() / (DataType)RAND_MAX;
    }
  }

  for(int i=0;i<numARows;i++) {
        for(int j=0;j<numBColumns;j++) {
          resultRef[i*numBColumns+j] = 0;
          for(int k=0;k<numAColumns;k++) {
            resultRef[i*numBColumns+j] += hostA[i*numAColumns+k] * hostB[k*numBColumns+j];
          }        
        }
  }

  //@@ Insert code below to allocate GPU memory here

  hipMalloc(&deviceA, numARows * numAColumns * sizeof(DataType));
  hipMalloc(&deviceB, numBRows * numBColumns * sizeof(DataType));
  hipMalloc(&deviceC, numCRows * numCColumns * sizeof(DataType));

  //@@ Insert code to below to Copy memory to the GPU here

  double start1 = timestart();
  hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(DataType), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(DataType), hipMemcpyHostToDevice);
  double time1 = timestop() - start1;
  printf("Transfer time host to device %f seconds\n", time1);

  //@@ Initialize the grid and block dimensions here

  int dimBlockx= TPB;
  int dimBlocky= TPB;

  int dimGridx= (numCColumns+dimBlockx-1)/dimBlockx;
  int dimGridy= (numCRows+dimBlocky-1)/dimBlocky;

  //@@ Launch the GPU Kernel here
  double start2 = timestart();
  gemm<<<dim3(dimGridx,dimGridy,1),dim3(dimBlockx,dimBlocky,1)>>>(deviceA,deviceB,deviceC,numARows,numAColumns,numBRows,numBColumns);
  hipDeviceSynchronize();
  double time2 = timestop() - start2;
  printf("Kernel Time %f seconds\n", time2);

  //@@ Copy the GPU memory back to the CPU here
  
  double start3=timestart();
  hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(DataType), hipMemcpyDeviceToHost);
  double time3 = timestop() - start3;
  printf("Transfer Time device to host %f seconds\n", time3);

  //@@ Insert code below to compare the output with the reference
 if (hostC == resultRef) printf("The output is same as the reference!\n");

  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  //@@ Free the CPU memory here
  free(hostA);
  free(hostB);
  free(hostC);
  free(resultRef);

  return 0;
}
