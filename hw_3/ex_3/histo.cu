

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <random>

#define NUM_BINS 1024
#define TPB1 32
#define TPB2 32

__global__ void histogram_kernel(unsigned int *input, unsigned int *bins,
                                 unsigned int num_elements,
                                 unsigned int num_bins) {

//@@ Insert code below to compute histogram of input using shared memory and atomics

  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  __shared__ unsigned int shared_bins[NUM_BINS];

   if(threadIdx.x == 0){
      for (int i = 0; i < NUM_BINS; i++)
          shared_bins[i] = 0;
    }

  __syncthreads();

    atomicAdd(&shared_bins[input[idx]], 1);

  __syncthreads();

  if(threadIdx.x == 0){
    for (int i = 0; i < NUM_BINS; i++){
      atomicAdd(&bins[i],shared_bins[i]);
    }
  }

}

__global__ void convert_kernel(unsigned int *bins, unsigned int num_bins) {

//@@ Insert code below to clean up bins that saturate at 127

  int bins_id = blockDim.x * blockIdx.x + threadIdx.x;

  if(bins_id > num_bins) return;

  if(bins[bins_id] > 127) bins[bins_id] = 127;

}


int main(int argc, char **argv) {
  
  int inputLength;
  unsigned int *hostInput;
  unsigned int *hostBins;
  unsigned int *resultRef;
  unsigned int *deviceInput;
  unsigned int *deviceBins;

  //@@ Insert code below to read in inputLength from args

  if(argc > 1) inputLength = atoi(argv[1]);
  printf("The input length is %d\n", inputLength);
  
  //@@ Insert code below to allocate Host memory for input and output

  hostBins = (unsigned int*)malloc(sizeof(unsigned int) * NUM_BINS);
  hostInput = (unsigned int*)malloc(sizeof(unsigned int) * inputLength);
  resultRef = (unsigned int*)malloc(sizeof(unsigned int) * NUM_BINS);
  
  //@@ Insert code below to initialize hostInput to random numbers whose values range from 0 to (NUM_BINS - 1)

  srand((unsigned)time(NULL));
  for (int i = 0; i < inputLength; i++){
    hostInput[i] = (unsigned int)rand() % (unsigned int)rand();
  }

  //@@ Insert code below to create reference result in CPU

  for (int i=0;i<inputLength;i++){
    resultRef[i] = 0;
  }

  for (int i=0;i<inputLength;i++){
    resultRef[hostInput[i]]=resultRef[hostInput[i]]+1;
  }

  for (int i=0;i<NUM_BINS;i++){
    if(resultRef[hostInput[i]]>127)
      resultRef[hostInput[i]]=127;
  }

  //@@ Insert code below to allocate GPU memory here

  hipMalloc(&deviceInput, sizeof(unsigned int) * inputLength);
  hipMalloc(&deviceBins, sizeof(unsigned int) * NUM_BINS);

  //@@ Insert code to Copy memory to the GPU here

  hipMemcpy(deviceInput, hostInput, sizeof(unsigned int) * inputLength, hipMemcpyHostToDevice);

  //@@ Insert code to initialize GPU results
  
  hipMemset(deviceBins, 0, sizeof(unsigned int) * NUM_BINS);

  //@@ Initialize the grid and block dimensions here

  dim3 dimBlock1(TPB1,1,1);
  dim3 dimGrid1((inputLength+TPB1-1)/TPB1,1,1);

  //@@ Launch the GPU Kernel here

  histogram_kernel<<<dimGrid1,dimBlock1>>>(deviceInput, deviceBins, inputLength, NUM_BINS);
  //@@ Initialize the second grid and block dimensions here

  dim3 dimBlock2(TPB2,1,1);
  dim3 dimGrid2((inputLength+TPB2-1)/TPB2,1,1);
  //@@ Launch the second GPU Kernel here

  convert_kernel<<<dimGrid2,dimBlock2>>>(deviceBins,NUM_BINS);
  //@@ Copy the GPU memory back to the CPU here

  hipMemcpy(hostBins, deviceBins, sizeof(unsigned int) * NUM_BINS, hipMemcpyDeviceToHost);
  //@@ Insert code below to compare the output with the reference

  for (int i=0;i<NUM_BINS;i++){
    printf("%d, ",hostBins[i]);
  }

  printf("END!\n");

  for (int i=0;i<NUM_BINS;i++){
    printf("%d, ",resultRef[i]);
  }

  if (hostBins == resultRef) printf("The output is same as the reference!\n");  
  //@@ Free the GPU memory here

  hipFree(deviceInput);
  hipFree(deviceBins);
  //@@ Free the CPU memory here

  free(hostInput);
  free(hostBins);
  free(resultRef);

  return 0;
}

