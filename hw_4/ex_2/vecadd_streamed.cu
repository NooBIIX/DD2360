#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

#define DataType double
#define TPB 1024

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
  //@@ Insert code to implement vector addition here
  int id = blockIdx.x * blockDim.x + threadIdx.x;

  if(id < len) out[id] = in1[id] + in2[id];
}

//@@ Insert code to implement timer start
double timestart(){
  struct timeval t_start;
  gettimeofday(&t_start, NULL);
  return (double) (1000000.0*(t_start.tv_sec) + t_start.tv_usec)/1000000.0;
}
//@@ Insert code to implement timer stop
double timestop(){
  struct timeval t_stop;
  gettimeofday(&t_stop, NULL);
  return (double) (1000000.0*(t_stop.tv_sec) + t_stop.tv_usec)/1000000.0;
}

int main(int argc, char **argv) {
  
  int inputLength;
  DataType *hostInput1;
  DataType *hostInput2;
  DataType *hostOutput;
  DataType *resultRef;
  DataType *deviceInput1;
  DataType *deviceInput2;
  DataType *deviceOutput;

  //@@ Insert code below to read in inputLength from args

  if(argc > 1) inputLength = atoi(argv[1]);

  printf("The input length is %d\n", inputLength);
  
  //@@ Insert code below to allocate Host memory for input and output
  size_t bytes = inputLength * sizeof(double);


  hostInput1 = (double *)malloc(bytes);
  hostInput2 = (double *)malloc(bytes);
  hostOutput = (double *)malloc(bytes);
  resultRef = (double *)malloc(bytes);


  //@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers, and create reference result in CPU
  srand((unsigned)time(NULL));
  for(int i = 0; i < inputLength - 1; i++){
    hostInput1[i] = ((double)rand()/(double)RAND_MAX);
    hostInput2[i] = ((double)rand()/(double)RAND_MAX);
    resultRef[i] = hostInput1[i] + hostInput2[i];
  }

  //@@ Insert code below to allocate GPU memory here
  hipMalloc(&deviceInput1, bytes);
  hipMalloc(&deviceInput2, bytes);
  hipMalloc(&deviceOutput, bytes);


  //@@ Insert code to below to Copy memory to the GPU here

  int num_str = 1024;
  int S_seg=(inputLength + num_str - 1) / num_str; 

  hipStream_t stream[num_str];

  for(int i=0;i<num_str;i++){
    hipStreamCreate(&stream[i]);
  }

  double start1 = timestart();

  for(int i=0;i<num_str;i++){
    hipMemcpyAsync(deviceInput1+i*S_seg, hostInput1+i*S_seg, S_seg * sizeof(DataType), hipMemcpyHostToDevice, stream[i]);
    hipMemcpyAsync(deviceInput2+i*S_seg, hostInput2+i*S_seg, S_seg * sizeof(DataType), hipMemcpyHostToDevice, stream[i]);
  }
  double t1me = timestop() - start1;
  printf("Transfer time host to device %f seconds\n", t1me);


  //@@ Initialize the 1D grid and block dimensions here
  int blockSize, gridSize;

  blockSize = TPB;
  gridSize = (inputLength+blockSize-1) / blockSize;
  //printf("gridSize is %d\n", gridSize);

  //@@ Launch the GPU Kernel here
  double start_kernel = timestart();
  for(int i=0;i<num_str;i++){ 
    vecAdd<<<gridSize, blockSize, 0, stream[i]>>>(deviceInput1+i*S_seg, deviceInput2+i*S_seg, deviceOutput+i*S_seg, S_seg);
  }
  
  //hipDeviceSynchronize();

  double time_kernel = timestop() - start_kernel;
  printf("Kernel time %f seconds\n", time_kernel);

  //@@ Copy the GPU memory back to the CPU here
  double start2 = timestart();

  for(int i=0;i<num_str;i++){
    hipMemcpyAsync(hostOutput+i*S_seg, deviceOutput+i*S_seg, S_seg * sizeof(DataType), hipMemcpyDeviceToHost, stream[i]);
  }

  double t2me = timestop() - start2;
  printf("Transfer time device to host %f seconds\n", t2me);


  for(int i = 0; i<num_str;i++) {
      hipStreamDestroy(stream[i]);
  }
  hipDeviceSynchronize();
  //@@ Insert code below to compare the output with the reference
  if (hostOutput == resultRef) printf("The output is same as the reference!\n");

  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  //@@ Free the CPU memory here
  free(hostInput1);
  free(hostInput2);
  free(hostOutput);
  free(resultRef);

  return 0;
}
